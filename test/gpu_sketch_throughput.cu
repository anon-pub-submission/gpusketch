#include "hip/hip_runtime.h"
#include <chrono>
#include <cmath>
#include <vector>

#include <sketch.h>
#include "../src/cuda_kernel.cu"

static size_t get_seed() {
  auto now = std::chrono::high_resolution_clock::now();
  return std::chrono::duration_cast<std::chrono::nanoseconds>(now.time_since_epoch()).count();
}

__global__ void gpuSketchTest_kernel(int num_device_blocks, node_id_t num_nodes, size_t num_updates, node_id_t *edgeUpdates, size_t num_buckets, Bucket* buckets, size_t num_columns, size_t bkt_per_col, size_t sketchSeed) {

  extern __shared__ vec_t_cu sketches[];
  vec_t_cu* bucket_a = sketches;
  vec_hash_t* bucket_c = (vec_hash_t*)&bucket_a[num_buckets];

  for (size_t i = threadIdx.x; i < num_buckets; i += blockDim.x) {
    bucket_a[i] = 0;
    bucket_c[i] = 0;
  }

  __syncthreads();

  node_id_t node_id = blockIdx.x / num_nodes;
  for (size_t id = threadIdx.x; id < num_updates * num_columns; id += blockDim.x) {

    size_t column_id = id % num_columns;
    size_t update_id = id / num_columns;

    // Get random edge id based on current update_id
    vec_t edge_id = device_concat_pairing_fn(node_id, edgeUpdates[update_id]);

    vec_hash_t checksum = bucket_get_index_hash(edge_id, sketchSeed);
    
    if ((blockIdx.x == num_device_blocks - 1)  && (column_id == 0)) {
      // Update depth 0 bucket
      bucket_update(bucket_a[num_buckets - 1], bucket_c[num_buckets - 1], edge_id, checksum);
    }

    // Update higher depth buckets
    col_hash_t depth = bucket_get_index_depth(edge_id, sketchSeed + ((column_id) * 5), bkt_per_col);
    size_t bucket_id = column_id * bkt_per_col + depth;
    if(depth < bkt_per_col)
      bucket_update(bucket_a[bucket_id], bucket_c[bucket_id], edge_id, checksum);
  }

  __syncthreads();

  for (size_t i = threadIdx.x; i < num_buckets; i += blockDim.x) {
    //atomicXor((vec_t_cu*)&buckets[(node_id * num_buckets) + i].alpha, bucket_a[i]);
    //atomicXor((vec_t_cu*)&buckets[(node_id * num_buckets) + i].gamma, (vec_t_cu)bucket_c[i]);
    buckets[(node_id * num_buckets) + i].alpha = bucket_a[i];
    buckets[(node_id * num_buckets) + i].gamma = bucket_c[i];
  }
}


int main(int argc, char **argv) {
  if (argc != 4) {
    std::cout << "ERROR: Incorrect number of arguments!" << std::endl;
    std::cout << "Arguments: num_nodes num_updates GPU_ID" << std::endl;
    exit(EXIT_FAILURE);
  }

  std::cout << "SKETCH COMPUTE THROUGHPUT TEST - GPU:\n";
  gpuErrchk(hipSetDevice(std::atoi(argv[3])));
  int device_id = hipGetDevice(&device_id);
  int device_count = 0;

  hipGetDeviceCount(&device_count);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, device_id);
  std::cout << "-----CUDA Device Information-----\n";
  std::cout << "CUDA Device Count: " << device_count << "\n";
  std::cout << "CUDA Device ID: " << device_id << "\n";
  std::cout << "CUDA Device Number of SMs: " << deviceProp.multiProcessorCount << "\n"; 
  std::cout << "CUDA Max. Shared memory per Block: " << (double)deviceProp.sharedMemPerBlockOptin / 1000 << "KB\n";

  size_t free_memory;
  size_t total_memory;

  hipMemGetInfo(&free_memory, &total_memory);
  std::cout << "GPU Free (Available) Memory: " << (double)free_memory / 1000000000 << "GB\n";
  std::cout << "GPU Total Memory: " << (double)total_memory / 1000000000 << "GB\n";
  std::cout << "\n";

  node_id_t num_nodes = std::atoi(argv[1]);
  size_t num_updates = std::stoull(argv[2]);

  // Single Sketch with size corresponding to num_nodes
  SketchParams sketchParams;
  sketchParams.num_samples = Sketch::calc_cc_samples(num_nodes, 1);
  sketchParams.num_columns = sketchParams.num_samples * Sketch::default_cols_per_sample;
  sketchParams.bkt_per_col = Sketch::calc_bkt_per_col(Sketch::calc_vector_length(num_nodes));
  sketchParams.num_buckets = sketchParams.num_columns * sketchParams.bkt_per_col + 1;

  std::cout << "-----Sketch Information-----\n";
  std::cout << "num_nodes: " << num_nodes << "\n";
  std::cout << "num_updates: " << num_updates << "\n";
  std::cout << "bkt_per_col: " << sketchParams.bkt_per_col << "\n";
  std::cout << "num_columns: " << sketchParams.num_columns << "\n";
  std::cout << "num_buckets: " << sketchParams.num_buckets << "\n";
  std::cout << "\n";

  int num_device_threads = 1024;
  size_t num_updates_per_blocks = (sketchParams.num_buckets * sizeof(Bucket)) / sizeof(node_id_t);
  size_t num_device_blocks = std::ceil(((double)num_updates * 2) / num_updates_per_blocks);

  std::cout << "Batch Size: " << num_updates_per_blocks << "\n\n";

  size_t maxBytes = (sketchParams.num_buckets * sizeof(vec_t_cu)) + (sketchParams.num_buckets * sizeof(vec_hash_t));
  hipFuncSetAttribute(reinterpret_cast<const void*>(gpuSketchTest_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, maxBytes);

  std::cout << "-----GPU Kernel Information-----\n";
  std::cout << "Number of thread blocks: " << num_device_blocks << "\n";
  std::cout << "Number of threads per block: " << num_device_threads << "\n";
  std::cout << "Memory Size for buckets: " << (double)(num_nodes * sketchParams.num_buckets * sizeof(Bucket)) / 1000000000 << "GB\n";
  std::cout << "  Allocated Shared Memory of: " << (double)maxBytes / 1000 << "KB\n";
  std::cout << "\n";

  Bucket* d_buckets;
  gpuErrchk(hipMalloc(&d_buckets, num_nodes * sketchParams.num_buckets * sizeof(Bucket)));

  node_id_t *h_edgeUpdates, *d_edgeUpdates;
  gpuErrchk(hipHostMalloc(&h_edgeUpdates, num_updates_per_blocks * sizeof(node_id_t)));
  gpuErrchk(hipMalloc(&d_edgeUpdates, num_updates_per_blocks * sizeof(node_id_t)));

  for (size_t update_id = 0; update_id < num_updates_per_blocks; update_id++) {
    h_edgeUpdates[update_id] = update_id;
  }
  
  gpuErrchk(hipMemcpy(d_edgeUpdates, h_edgeUpdates, num_updates_per_blocks * sizeof(node_id_t), hipMemcpyHostToDevice));

  size_t sketchSeed = get_seed();

  float time;
  hipEvent_t start, stop;

  gpuErrchk(hipEventCreate(&start));
  gpuErrchk(hipEventCreate(&stop));

  auto sketch_update_start = std::chrono::steady_clock::now();
  gpuErrchk(hipEventRecord(start));
  gpuSketchTest_kernel<<<num_device_blocks, num_device_threads, maxBytes>>>(num_device_blocks, num_nodes, num_updates_per_blocks, d_edgeUpdates, sketchParams.num_buckets, d_buckets, sketchParams.num_columns, sketchParams.bkt_per_col, sketchSeed);
  gpuErrchk(hipEventRecord(stop));

  hipDeviceSynchronize();
  std::chrono::duration<double> sketch_update_duration = std::chrono::steady_clock::now() - sketch_update_start;

  gpuErrchk(hipEventSynchronize(stop));
  gpuErrchk(hipEventElapsedTime(&time, start, stop));
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) 
      printf("Error: %s\n", hipGetErrorString(err));

  std::cout << "Device Sync + CPU - Kernel Execution Time (s):    " << sketch_update_duration.count() << std::endl;
  std::cout << "Device Sync + CPU - Rate (# of Edges / s):        " << num_updates / sketch_update_duration.count() << std::endl;
  std::cout << "CUDA Event - Kernel Execution Time (s):           " << time * 0.001 << std::endl;
  std::cout << "CUDA Event - Rate (# of Edges / s):               " << num_updates / (time * 0.001) << std::endl;

  hipFree(d_buckets);
}
