#include "hip/hip_runtime.h"
#include <chrono>
#include <cmath>
#include <vector>

#include <sketch.h>
#include "../src/cuda_kernel.cu"

static size_t get_seed() {
  auto now = std::chrono::high_resolution_clock::now();
  return std::chrono::duration_cast<std::chrono::nanoseconds>(now.time_since_epoch()).count();
}

__global__ void gpuSketchTest_kernel(int num_device_blocks, node_id_t num_nodes, size_t num_updates, node_id_t *edgeUpdates, size_t num_buckets, Bucket* buckets, size_t num_columns, size_t bkt_per_col, size_t sketchSeed) {

  extern __shared__ vec_t_cu sketches[];
  vec_t_cu* bucket_a = sketches;
  vec_hash_t* bucket_c = (vec_hash_t*)&bucket_a[num_buckets];

  for (size_t i = threadIdx.x; i < num_buckets; i += blockDim.x) {
    bucket_a[i] = 0;
    bucket_c[i] = 0;
  }

  __syncthreads();

  node_id_t node_id = blockIdx.x / num_nodes;
  for (size_t id = threadIdx.x; id < num_updates * num_columns; id += blockDim.x) {

    size_t column_id = id % num_columns;
    size_t update_id = id / num_columns;

    // Get random edge id based on current update_id
    vec_t edge_id = device_concat_pairing_fn(node_id, edgeUpdates[update_id]);

    vec_hash_t checksum = bucket_get_index_hash(edge_id, sketchSeed);
    
    if ((blockIdx.x == num_device_blocks - 1)  && (column_id == 0)) {
      // Update depth 0 bucket
      bucket_update(bucket_a[num_buckets - 1], bucket_c[num_buckets - 1], edge_id, checksum);
    }

    // Update higher depth buckets
    col_hash_t depth = bucket_get_index_depth(edge_id, sketchSeed + ((column_id) * 5), bkt_per_col);
    size_t bucket_id = column_id * bkt_per_col + depth;
    if(depth < bkt_per_col)
      bucket_update(bucket_a[bucket_id], bucket_c[bucket_id], edge_id, checksum);
  }

  __syncthreads();

  for (size_t i = threadIdx.x; i < num_buckets; i += blockDim.x) {
    buckets[(node_id * num_buckets) + i].alpha = bucket_a[i];
    buckets[(node_id * num_buckets) + i].gamma = bucket_c[i];
  }
}


int main(int argc, char **argv) {
  if (argc != 5) {
    std::cout << "ERROR: Incorrect number of arguments!" << std::endl;
    std::cout << "Arguments: num_nodes start_density max_density density_inc" << std::endl;
    exit(EXIT_FAILURE);
  }

  int device_id = hipGetDevice(&device_id);
  int device_count = 0;

  hipGetDeviceCount(&device_count);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, device_id);
  std::cout << "-----CUDA Device Information-----\n";
  std::cout << "CUDA Device Count: " << device_count << "\n";
  std::cout << "CUDA Device ID: " << device_id << "\n";
  std::cout << "CUDA Device Number of SMs: " << deviceProp.multiProcessorCount << "\n"; 
  std::cout << "CUDA Max. Shared memory per Block: " << (double)deviceProp.sharedMemPerBlockOptin / 1000 << "KB\n";

  size_t free_memory;
  size_t total_memory;

  hipMemGetInfo(&free_memory, &total_memory);
  std::cout << "GPU Free (Available) Memory: " << (double)free_memory / 1000000000 << "GB\n";
  std::cout << "GPU Total Memory: " << (double)total_memory / 1000000000 << "GB\n";
  std::cout << "\n";

  node_id_t num_nodes = std::atoi(argv[1]);
  double start_density = std::stod(argv[2]);
  double max_density = std::stod(argv[3]);
  double density_inc = std::stod(argv[4]);
  size_t num_complete_edges = (((size_t)num_nodes * ((size_t)num_nodes - 1)) / 2);

  std::cout << "Max Density: " << max_density * 100 << "%\n";

  // Single Sketch with size corresponding to num_nodes
  SketchParams sketchParams;
  sketchParams.num_samples = Sketch::calc_cc_samples(num_nodes, 1);
  sketchParams.num_columns = sketchParams.num_samples * Sketch::default_cols_per_sample;
  sketchParams.bkt_per_col = Sketch::calc_bkt_per_col(Sketch::calc_vector_length(num_nodes));
  sketchParams.num_buckets = sketchParams.num_columns * sketchParams.bkt_per_col + 1;

  std::cout << "-----Sketch Information-----\n";
  std::cout << "num_nodes: " << num_nodes << "\n";
  std::cout << "num_complete_edges: " << num_complete_edges << "\n";
  std::cout << "bkt_per_col: " << sketchParams.bkt_per_col << "\n";
  std::cout << "num_columns: " << sketchParams.num_columns << "\n";
  std::cout << "num_buckets: " << sketchParams.num_buckets << "\n";
  std::cout << "\n";

  int num_device_threads = 1024;
  size_t num_updates_per_blocks = (sketchParams.num_buckets * sizeof(Bucket)) / sizeof(node_id_t);
  size_t num_max_device_blocks = std::ceil(((double)num_complete_edges * 2) / num_updates_per_blocks);

  std::cout << "Batch Size: " << num_updates_per_blocks << "\n\n";

  size_t maxBytes = (sketchParams.num_buckets * sizeof(vec_t_cu)) + (sketchParams.num_buckets * sizeof(vec_hash_t));
  hipFuncSetAttribute(reinterpret_cast<const void*>(gpuSketchTest_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, maxBytes);

  std::cout << "-----GPU Kernel Information-----\n";
  std::cout << "Number of max thread blocks: " << num_max_device_blocks << "\n";
  std::cout << "Number of threads per block: " << num_device_threads << "\n";
  std::cout << "Memory Size for buckets: " << (double)(num_nodes * sketchParams.num_buckets * sizeof(Bucket)) / 1000000000 << "GB\n";
  std::cout << "  Allocated Shared Memory of: " << (double)maxBytes / 1000 << "KB\n";
  std::cout << "\n";

  std::cout << "Allocating Host Memory: " << (num_updates_per_blocks * sizeof(node_id_t)) / 1e9 << "GB\n";
  std::cout << "Allocating GPU Memory: " << ((num_nodes * sketchParams.num_buckets * sizeof(Bucket)) + (num_updates_per_blocks * sizeof(node_id_t))) / 1e9 << "GB\n";

  Bucket* d_buckets;
  gpuErrchk(hipMalloc(&d_buckets, num_nodes * sketchParams.num_buckets * sizeof(Bucket)));

  node_id_t *h_edgeUpdates, *d_edgeUpdates;

  gpuErrchk(hipHostMalloc(&h_edgeUpdates, num_updates_per_blocks * sizeof(node_id_t)));
  gpuErrchk(hipMalloc(&d_edgeUpdates, num_updates_per_blocks * sizeof(node_id_t)));

  for (size_t update_id = 0; update_id < num_updates_per_blocks; update_id++) {
    h_edgeUpdates[update_id] = update_id;
  }
  
  gpuErrchk(hipMemcpy(d_edgeUpdates, h_edgeUpdates, num_updates_per_blocks * sizeof(node_id_t), hipMemcpyHostToDevice));

  size_t sketchSeed = get_seed();

  float time;
  hipEvent_t start, stop;

  gpuErrchk(hipEventCreate(&start));
  gpuErrchk(hipEventCreate(&stop));

  std::vector<double> densities = {0.0000001, 0.0000002, 0.0000003, 0.0000004, 0.0000005,
                                 0.0000006, 0.0000007, 0.0000008, 0.0000009, 0.000001, 
                                 0.000002, 0.000003, 0.000004, 0.000005,
                                 0.000006, 0.000007, 0.000008, 0.000009, 0.00001, 
                                 0.00002, 0.00003, 0.00004, 0.00005, 0.00006, 
                                 0.00007, 0.00008, 0.00009, 0.0001, 0.0002, 0.0003, 
                                 0.0004, 0.0005, 0.0006, 0.0007, 0.0008, 0.0009,
                                 0.001, 0.002, 0.003, 0.004, 0.005, 0.006, 0.007, 
                                 0.008, 0.009, 0.01, 0.02, 0.03, 0.04, 0.05, 0.06, 
                                 0.07, 0.08, 0.09, 0.1, 0.15, 0.2};

  for (auto& density : densities) {                                
  //for (double density = start_density; density < (max_density + 0.000001); density += density_inc) {
    int num_device_blocks = density * num_max_device_blocks;

    std::cout << "Density: " << density * 100 << "%\n";
    std::cout << "  Number of device blocks: " << num_device_blocks << "\n";
    std::cout << "  Number of updates: " << num_updates_per_blocks * num_device_blocks << "\n";

    if (num_device_blocks == 0) {
      std::cout << "  Current Density too low, skipping\n";
      continue;
    }

    gpuErrchk(hipEventRecord(start));
    gpuSketchTest_kernel<<<num_device_blocks, num_device_threads, maxBytes>>>(num_device_blocks, num_nodes, num_updates_per_blocks, d_edgeUpdates, sketchParams.num_buckets, d_buckets, sketchParams.num_columns, sketchParams.bkt_per_col, sketchSeed);
    gpuErrchk(hipEventRecord(stop));

    hipDeviceSynchronize();

    gpuErrchk(hipEventSynchronize(stop));
    gpuErrchk(hipEventElapsedTime(&time, start, stop));
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));

    std::cout << "CUDA Event - Kernel Execution Time (s):           " << time * 0.001 << std::endl;
    std::cout << "CUDA Event - Rate (# of Edges / s):               " << ((num_updates_per_blocks * num_device_blocks) / 2) / (time * 0.001) << std::endl;
  } 

  hipFree(d_buckets);
}
