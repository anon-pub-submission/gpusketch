#include "hip/hip_runtime.h"
#include <chrono>
#include <cmath>
#include <vector>

#include <sketch.h>
#include "../src/cuda_kernel.cu"

static size_t get_seed() {
  auto now = std::chrono::high_resolution_clock::now();
  return std::chrono::duration_cast<std::chrono::nanoseconds>(now.time_since_epoch()).count();
}

__global__ void gpuSketchTest_kernel(int num_device_blocks, node_id_t num_nodes, size_t num_updates_per_block, node_id_t *edgeUpdates, size_t num_buckets, Bucket* buckets, size_t num_columns, size_t bkt_per_col, size_t sketchSeed) {

  extern __shared__ vec_t_cu sketches[];
  vec_t_cu* bucket_a = sketches;
  vec_hash_t* bucket_c = (vec_hash_t*)&bucket_a[num_buckets];

  for (size_t i = threadIdx.x; i < num_buckets; i += blockDim.x) {
    bucket_a[i] = 0;
    bucket_c[i] = 0;
  }

  __syncthreads();

  node_id_t node_id = blockIdx.x % num_nodes;
  for (size_t id = threadIdx.x; id < num_updates_per_block * num_columns; id += blockDim.x) {

    size_t column_id = id % num_columns;
    size_t update_id = id / num_columns;

    // Get random edge id based on current update_id
    vec_t edge_id = device_concat_pairing_fn(node_id, edgeUpdates[(blockIdx.x * num_updates_per_block) + update_id]);

    vec_hash_t checksum = bucket_get_index_hash(edge_id, sketchSeed);
    
    if ((blockIdx.x == num_device_blocks - 1)  && (column_id == 0)) {
      // Update depth 0 bucket
      bucket_update(bucket_a[num_buckets - 1], bucket_c[num_buckets - 1], edge_id, checksum);
    }

    // Update higher depth buckets
    col_hash_t depth = bucket_get_index_depth(edge_id, sketchSeed + ((column_id) * 5), bkt_per_col);
    size_t bucket_id = column_id * bkt_per_col + depth;
    if(depth < bkt_per_col)
      bucket_update(bucket_a[bucket_id], bucket_c[bucket_id], edge_id, checksum);
  }

  __syncthreads();

  for (size_t i = threadIdx.x; i < num_buckets; i += blockDim.x) {
    atomicXor((vec_t_cu*)&buckets[(node_id * num_buckets) + i].alpha, bucket_a[i]);
    atomicXor((vec_t_cu*)&buckets[(node_id * num_buckets) + i].gamma, (vec_t_cu)bucket_c[i]);
  }
}


int main(int argc, char **argv) {
  if (argc != 2) {
    std::cout << "ERROR: Incorrect number of arguments!" << std::endl;
    std::cout << "Arguments: num_nodes" << std::endl;
    exit(EXIT_FAILURE);
  }

  int device_id = hipGetDevice(&device_id);
  int device_count = 0;

  hipGetDeviceCount(&device_count);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, device_id);
  std::cout << "-----CUDA Device Information-----\n";
  std::cout << "CUDA Device Count: " << device_count << "\n";
  std::cout << "CUDA Device ID: " << device_id << "\n";
  std::cout << "CUDA Device Number of SMs: " << deviceProp.multiProcessorCount << "\n"; 
  std::cout << "CUDA Max. Shared memory per Block: " << (double)deviceProp.sharedMemPerBlockOptin / 1000 << "KB\n";

  size_t free_memory;
  size_t total_memory;

  hipMemGetInfo(&free_memory, &total_memory);
  std::cout << "GPU Free (Available) Memory: " << (double)free_memory / 1000000000 << "GB\n";
  std::cout << "GPU Total Memory: " << (double)total_memory / 1000000000 << "GB\n";
  std::cout << "\n";

  node_id_t num_nodes = std::atoi(argv[1]);

  // Single Sketch with size corresponding to num_nodes
  SketchParams sketchParams;
  sketchParams.num_samples = Sketch::calc_cc_samples(num_nodes, 1);
  sketchParams.num_columns = sketchParams.num_samples * Sketch::default_cols_per_sample;
  sketchParams.bkt_per_col = Sketch::calc_bkt_per_col(Sketch::calc_vector_length(num_nodes));
  sketchParams.num_buckets = sketchParams.num_columns * sketchParams.bkt_per_col + 1;

  std::cout << "-----Sketch Information-----\n";
  std::cout << "num_nodes: " << num_nodes << "\n";
  std::cout << "bkt_per_col: " << sketchParams.bkt_per_col << "\n";
  std::cout << "num_columns: " << sketchParams.num_columns << "\n";
  std::cout << "num_buckets: " << sketchParams.num_buckets << "\n";
  std::cout << "\n";

  int num_device_threads = 1024;
  size_t num_updates_per_block = (sketchParams.num_buckets * sizeof(Bucket)) / sizeof(node_id_t);

  std::cout << "Batch Size: " << num_updates_per_block << "\n\n";

  size_t maxBytes = (sketchParams.num_buckets * sizeof(vec_t_cu)) + (sketchParams.num_buckets * sizeof(vec_hash_t));
  hipFuncSetAttribute(reinterpret_cast<const void*>(gpuSketchTest_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, maxBytes);

  std::cout << "-----GPU Kernel Information-----\n";
  std::cout << "Number of threads per block: " << num_device_threads << "\n";
  std::cout << "Memory Size for buckets: " << (double)(num_nodes * sketchParams.num_buckets * sizeof(Bucket)) / 1000000000 << "GB\n";
  std::cout << "  Allocated Shared Memory of: " << (double)maxBytes / 1000 << "KB\n";
  std::cout << "\n";

  Bucket* d_buckets;
  gpuErrchk(hipMalloc(&d_buckets, num_nodes * sketchParams.num_buckets * sizeof(Bucket)));

  node_id_t *h_edgeUpdates, *d_edgeUpdates;
  size_t max_num_updates = 4e9;
  size_t max_num_blocks = (2 * max_num_updates) / num_updates_per_block;
  size_t max_act_updates = max_num_blocks * num_updates_per_block;

  gpuErrchk(hipHostMalloc(&h_edgeUpdates, max_act_updates * sizeof(node_id_t)));
  gpuErrchk(hipMalloc(&d_edgeUpdates, max_act_updates * sizeof(node_id_t)));

  std::cout << "Allocating Host Memory: " << (max_act_updates * sizeof(node_id_t)) / 1e9 << "GB\n";
  std::cout << "Allocating GPU Memory: " << ((num_nodes * sketchParams.num_buckets * sizeof(Bucket)) + (max_act_updates * sizeof(node_id_t))) / 1e9 << "GB\n";

  for (size_t block_id = 0; block_id < max_num_blocks; block_id++) {
    for (size_t update_id = 0; update_id < num_updates_per_block; update_id++) {
      h_edgeUpdates[(block_id * num_updates_per_block) + update_id] = update_id;
    }
  }

  gpuErrchk(hipMemcpy(d_edgeUpdates, h_edgeUpdates, max_act_updates * sizeof(node_id_t), hipMemcpyHostToDevice));

  size_t sketchSeed = get_seed();

  float time;
  hipEvent_t start, stop;

  gpuErrchk(hipEventCreate(&start));
  gpuErrchk(hipEventCreate(&stop));

  std::vector<double> stream_updates = {10000, 20000, 30000, 40000, 50000,
                                        60000, 70000, 80000, 90000, 100000,
                                        200000, 300000, 400000, 500000,
                                        600000, 700000, 800000, 900000, 1000000,
                                        2000000, 3000000, 4000000, 5000000,
                                        6000000, 7000000, 8000000, 9000000, 10000000,
                                        20000000, 30000000, 40000000, 50000000,
                                        60000000, 70000000, 80000000, 90000000, 100000000,
                                        200000000, 300000000, 400000000, 500000000,
                                        600000000, 700000000, 800000000, 900000000, 1000000000,
                                        2000000000, 3000000000, 4000000000};

  for (auto& stream_update : stream_updates) {                                
    size_t num_device_blocks = (2 * stream_update) / num_updates_per_block;

    std::cout << "Number of stream updates: " << stream_update << "\n";
    std::cout << "  Number of device blocks: " << num_device_blocks << "\n";
    std::cout << "  Number of updates: " << num_updates_per_block * num_device_blocks << "\n";

    if (num_device_blocks == 0) {
      std::cout << "  Current number of stream updates too low, skipping\n";
      continue;
    }

    if (num_device_blocks > max_num_blocks) {
      std::cout << "  Current number of stream updates exceeds maximum, breaking out\n";
      break;
    }

    gpuErrchk(hipEventRecord(start));
    gpuSketchTest_kernel<<<num_device_blocks, num_device_threads, maxBytes>>>(num_device_blocks, num_nodes, num_updates_per_block, d_edgeUpdates, sketchParams.num_buckets, d_buckets, sketchParams.num_columns, sketchParams.bkt_per_col, sketchSeed);
    gpuErrchk(hipEventRecord(stop));

    hipDeviceSynchronize();

    gpuErrchk(hipEventSynchronize(stop));
    gpuErrchk(hipEventElapsedTime(&time, start, stop));
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));

    std::cout << "CUDA Event - Kernel Execution Time (s):           " << time * 0.001 << std::endl;
    std::cout << "CUDA Event - Rate (# of Edges / s):               " << ((num_updates_per_block * num_device_blocks) / 2) / (time * 0.001) << std::endl;
  } 

  hipFree(d_buckets);
}
